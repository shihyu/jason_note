#include "hip/hip_runtime.h"
#include <cstdio>
#include <vector>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#include "hip/hip_vector_types.h"
#include "CudaArray.cuh"
#include "ticktock.h"
#include "writevdb.h"
#include <thread>

__global__ void advect_kernel(CudaTextureAccessor<float4> texVel, CudaSurfaceAccessor<float4> sufLoc, unsigned int n) {
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int z = threadIdx.z + blockDim.z * blockIdx.z;
    if (x >= n || y >= n || z >= n) return;

    auto sample = [] (CudaTextureAccessor<float4> tex, float3 loc) -> float3 {
        float4 vel = tex.sample(loc.x, loc.y, loc.z);
        return make_float3(vel.x, vel.y, vel.z);
    };

    float3 loc = make_float3(x + 0.5f, y + 0.5f, z + 0.5f);
    float3 vel1 = sample(texVel, loc);
    float3 vel2 = sample(texVel, loc - 0.5f * vel1);
    float3 vel3 = sample(texVel, loc - 0.75f * vel2);
    loc -= (2.f / 9.f) * vel1 + (1.f / 3.f) * vel2 + (4.f / 9.f) * vel3;
    sufLoc.write(make_float4(loc.x, loc.y, loc.z, 0.f), x, y, z);
}

__global__ void resample_kernel(CudaSurfaceAccessor<float4> sufLoc, CudaTextureAccessor<float4> texClr, CudaSurfaceAccessor<float4> sufClrNext, unsigned int n) {
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int z = threadIdx.z + blockDim.z * blockIdx.z;
    if (x >= n || y >= n || z >= n) return;

    float4 loc = sufLoc.read(x, y, z);
    float4 clr = texClr.sample(loc.x, loc.y, loc.z);
    sufClrNext.write(clr, x, y, z);
}

__global__ void divergence_kernel(CudaSurfaceAccessor<float4> sufVel, CudaSurfaceAccessor<float> sufDiv, unsigned int n) {
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int z = threadIdx.z + blockDim.z * blockIdx.z;
    if (x >= n || y >= n || z >= n) return;

    float4 vel = sufVel.read(x, y, z);
    float vxn = sufVel.read<hipBoundaryModeZero>(x - 1, y, z).x;
    float vyn = sufVel.read<hipBoundaryModeZero>(x, y - 1, z).y;
    float vzn = sufVel.read<hipBoundaryModeZero>(x, y, z - 1).z;
    float div = (vel.x - vxn + vel.y - vyn + vel.z - vzn);
    sufDiv.write(div, x, y, z);
}

/*__global__ void vorticity_kernel(CudaSurfaceAccessor<float4> sufVel, CudaSurfaceAccessor<float> sufVor, unsigned int n) {
    unsigned int x = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int y = threadIdx.y + blockDim.y * blockIdx.y;
    unsigned int z = threadIdx.z + blockDim.z * blockIdx.z;
    if (x >= n || y >= n || z >= n) return;

    float4 vel = sufVel.read<hipBoundaryModeClamp>(x, y, z);
    float4 vxp = sufVel.read<hipBoundaryModeClamp>(x + 1, y, z);
    float4 vyp = sufVel.read<hipBoundaryModeClamp>(x, y + 1, z);
    float4 vzp = sufVel.read<hipBoundaryModeClamp>(x, y, z + 1);
    float vorx = vel.y - vel.z - vzp.y + vyp.z;
    float vory = vel.z - vel.x - vxp.z + vzp.x;
    float vorz = vel.x - vel.y - vyp.x + vxp.y;
    float vor = sqrtf(vorx * vorx + vory * vory + vorz * vorz); // make_float4(vorx, vory, vorz, 0.f);
    sufVor.write(vor, x, y, z);
}

__global__ void enhancevor_kernel(CudaSurfaceAccessor<float4> sufVel, CudaSurfaceAccessor<float> sufVor, unsigned int n) {
    unsigned int x = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int y = threadIdx.y + blockDim.y * blockIdx.y;
    unsigned int z = threadIdx.z + blockDim.z * blockIdx.z;
    if (x >= n || y >= n || z >= n) return;

    float4 vel = sufVel.read<hipBoundaryModeClamp>(x, y, z);
    float4 vxp = sufVel.read<hipBoundaryModeClamp>(x + 1, y, z);
    float4 vyp = sufVel.read<hipBoundaryModeClamp>(x, y + 1, z);
    float4 vzp = sufVel.read<hipBoundaryModeClamp>(x, y, z + 1);
    float vorx = vel.y - vel.z - vzp.y + vyp.z;
    float vory = vel.z - vel.x - vxp.z + vzp.x;
    float vorz = vel.x - vel.y - vyp.x + vxp.y;
    float4 vor = make_float4(vorx, vory, vorz, 0.f);
    sufVor.write(vor, x, y, z);
}*/

__global__ void sumloss_kernel(CudaSurfaceAccessor<float> sufDiv, float *sum, unsigned int n) {
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int z = threadIdx.z + blockDim.z * blockIdx.z;
    if (x >= n || y >= n || z >= n) return;

    float div = sufDiv.read(x, y, z);
    atomicAdd(sum, div * div);
}

/*__global__ void jacobi_kernel(CudaSurfaceAccessor<float> sufDiv, CudaSurfaceAccessor<float> sufPre, CudaSurfaceAccessor<float> sufPreNext, unsigned int n) {
    unsigned int x = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int y = threadIdx.y + blockDim.y * blockIdx.y;
    unsigned int z = threadIdx.z + blockDim.z * blockIdx.z;
    if (x >= n || y >= n || z >= n) return;

    float pxp = sufPre.read<hipBoundaryModeClamp>(x + 1, y, z);
    float pxn = sufPre.read<hipBoundaryModeClamp>(x - 1, y, z);
    float pyp = sufPre.read<hipBoundaryModeClamp>(x, y + 1, z);
    float pyn = sufPre.read<hipBoundaryModeClamp>(x, y - 1, z);
    float pzp = sufPre.read<hipBoundaryModeClamp>(x, y, z + 1);
    float pzn = sufPre.read<hipBoundaryModeClamp>(x, y, z - 1);
    float div = sufDiv.read(x, y, z);
    float preNext = (pxp + pxn + pyp + pyn + pzp + pzn - div) * (1.f / 6.f);
    sufPreNext.write(preNext, x, y, z);
}*/

__global__ void subgradient_kernel(CudaSurfaceAccessor<float> sufPre, CudaSurfaceAccessor<float4> sufVel, unsigned int n) {
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int z = threadIdx.z + blockDim.z * blockIdx.z;
    if (x >= n || y >= n || z >= n) return;

    float pre = sufPre.read(x, y, z);
    float pxp = sufPre.read<hipBoundaryModeClamp>(x + 1, y, z);
    float pyp = sufPre.read<hipBoundaryModeClamp>(x, y + 1, z);
    float pzp = sufPre.read<hipBoundaryModeClamp>(x, y, z + 1);
    float4 vel = sufVel.read(x, y, z);
    vel.x -= (pxp - pre);
    vel.y -= (pyp - pre);
    vel.z -= (pzp - pre);
    sufVel.write(vel, x, y, z);
}

template <int phase>
__global__ void rbgs_kernel(CudaSurfaceAccessor<float> sufPre, CudaSurfaceAccessor<float> sufDiv, unsigned int n) {
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int z = threadIdx.z + blockDim.z * blockIdx.z;
    if (x >= n || y >= n || z >= n) return;
    if ((x + y + z) % 2 != phase) return;

    float pxp = sufPre.read<hipBoundaryModeClamp>(x + 1, y, z);
    float pxn = sufPre.read<hipBoundaryModeClamp>(x - 1, y, z);
    float pyp = sufPre.read<hipBoundaryModeClamp>(x, y + 1, z);
    float pyn = sufPre.read<hipBoundaryModeClamp>(x, y - 1, z);
    float pzp = sufPre.read<hipBoundaryModeClamp>(x, y, z + 1);
    float pzn = sufPre.read<hipBoundaryModeClamp>(x, y, z - 1);
    float div = sufDiv.read(x, y, z);
    float preNext = (pxp + pxn + pyp + pyn + pzp + pzn - div) * (1.f / 6.f);
    sufPre.write(preNext, x, y, z);
}

__global__ void residual_kernel(CudaSurfaceAccessor<float> sufRes, CudaSurfaceAccessor<float> sufPre, CudaSurfaceAccessor<float> sufDiv, unsigned int n) {
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int z = threadIdx.z + blockDim.z * blockIdx.z;
    if (x >= n || y >= n || z >= n) return;

    float pxp = sufPre.read<hipBoundaryModeClamp>(x + 1, y, z);
    float pxn = sufPre.read<hipBoundaryModeClamp>(x - 1, y, z);
    float pyp = sufPre.read<hipBoundaryModeClamp>(x, y + 1, z);
    float pyn = sufPre.read<hipBoundaryModeClamp>(x, y - 1, z);
    float pzp = sufPre.read<hipBoundaryModeClamp>(x, y, z + 1);
    float pzn = sufPre.read<hipBoundaryModeClamp>(x, y, z - 1);
    float pre = sufPre.read(x, y, z);
    float div = sufDiv.read(x, y, z);
    float res = pxp + pxn + pyp + pyn + pzp + pzn - 6.f * pre - div;
    sufRes.write(res, x, y, z);
}

__global__ void restrict_kernel(CudaSurfaceAccessor<float> sufPreNext, CudaSurfaceAccessor<float> sufPre, unsigned int n) {
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int z = threadIdx.z + blockDim.z * blockIdx.z;
    if (x >= n || y >= n || z >= n) return;

    float ooo = sufPre.read<hipBoundaryModeClamp>(x*2, y*2, z*2);
    float ioo = sufPre.read<hipBoundaryModeClamp>(x*2+1, y*2, z*2);
    float oio = sufPre.read<hipBoundaryModeClamp>(x*2, y*2+1, z*2);
    float iio = sufPre.read<hipBoundaryModeClamp>(x*2+1, y*2+1, z*2);
    float ooi = sufPre.read<hipBoundaryModeClamp>(x*2, y*2, z*2+1);
    float ioi = sufPre.read<hipBoundaryModeClamp>(x*2+1, y*2, z*2+1);
    float oii = sufPre.read<hipBoundaryModeClamp>(x*2, y*2+1, z*2+1);
    float iii = sufPre.read<hipBoundaryModeClamp>(x*2+1, y*2+1, z*2+1);
    float preNext = (ooo + ioo + oio + iio + ooi + ioi + oii + iii);
    sufPreNext.write(preNext, x, y, z);
}

__global__ void fillzero_kernel(CudaSurfaceAccessor<float> sufPre, unsigned int n) {
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int z = threadIdx.z + blockDim.z * blockIdx.z;
    if (x >= n || y >= n || z >= n) return;

    sufPre.write(0.f, x, y, z);
}

__global__ void prolongate_kernel(CudaSurfaceAccessor<float> sufPreNext, CudaSurfaceAccessor<float> sufPre, unsigned int n) {
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int z = threadIdx.z + blockDim.z * blockIdx.z;
    if (x >= n || y >= n || z >= n) return;

    float preDelta = sufPre.read(x, y, z) * (0.5f / 8.f);
#pragma unroll
    for (int dz = 0; dz < 2; dz++) {
#pragma unroll
        for (int dy = 0; dy < 2; dy++) {
#pragma unroll
            for (int dx = 0; dx < 2; dx++) {
                float preNext = sufPreNext.read<hipBoundaryModeZero>(x*2+dx, y*2+dy, z*2+dz);
                preNext += preDelta;
                sufPreNext.write<hipBoundaryModeZero>(preNext, x*2+dx, y*2+dy, z*2+dz);
            }
        }
    }
}

struct SmokeSim : DisableCopy {
    unsigned int n;
    std::unique_ptr<CudaSurface<float4>> loc;
    std::unique_ptr<CudaTexture<float4>> vel;
    std::unique_ptr<CudaTexture<float4>> velNext;
    std::unique_ptr<CudaTexture<float4>> clr;
    std::unique_ptr<CudaTexture<float4>> clrNext;

    std::unique_ptr<CudaSurface<float>> div;
    std::unique_ptr<CudaSurface<float>> pre;
    std::vector<std::unique_ptr<CudaSurface<float>>> res;
    std::vector<std::unique_ptr<CudaSurface<float>>> res2;
    std::vector<std::unique_ptr<CudaSurface<float>>> err2;
    std::vector<unsigned int> sizes;

    explicit SmokeSim(unsigned int _n, unsigned int _n0 = 16)
    : n(_n)
    , loc(std::make_unique<CudaSurface<float4>>(dim3{n, n, n}))
    , vel(std::make_unique<CudaTexture<float4>>(dim3{n, n, n}))
    , velNext(std::make_unique<CudaTexture<float4>>(dim3{n, n, n}))
    , clr(std::make_unique<CudaTexture<float4>>(dim3{n, n, n}))
    , clrNext(std::make_unique<CudaTexture<float4>>(dim3{n, n, n}))
    , div(std::make_unique<CudaSurface<float>>(dim3{n, n, n}))
    , pre(std::make_unique<CudaSurface<float>>(dim3{n, n, n}))
    {
        unsigned int tn;
        for (tn = n; tn >= _n0; tn /= 2) {
            res.push_back(std::make_unique<CudaSurface<float>>(dim3{tn, tn, tn}));
            res2.push_back(std::make_unique<CudaSurface<float>>(dim3{tn/2, tn/2, tn/2}));
            err2.push_back(std::make_unique<CudaSurface<float>>(dim3{tn/2, tn/2, tn/2}));
            sizes.push_back(tn);
        }
    }

    void smooth(CudaSurface<float> *v, CudaSurface<float> *f, unsigned int lev, int times = 4) {
        unsigned int tn = sizes[lev];
        for (int step = 0; step < times; step++) {
            rbgs_kernel<0><<<dim3((tn + 7) / 8, (tn + 7) / 8, (tn + 7) / 8), dim3(8, 8, 8)>>>(v->accessSurface(), f->accessSurface(), tn);
            rbgs_kernel<1><<<dim3((tn + 7) / 8, (tn + 7) / 8, (tn + 7) / 8), dim3(8, 8, 8)>>>(v->accessSurface(), f->accessSurface(), tn);
        }
    }

    void vcycle(unsigned int lev, CudaSurface<float> *v, CudaSurface<float> *f) {
        if (lev >= sizes.size()) {
            unsigned int tn = sizes.back() / 2;
            smooth(v, f, lev);
            return;
        }
        auto *r = res[lev].get();
        auto *r2 = res2[lev].get();
        auto *e2 = err2[lev].get();
        unsigned int tn = sizes[lev];
        smooth(v, f, lev);
        residual_kernel<<<dim3((tn + 7) / 8, (tn + 7) / 8, (tn + 7) / 8), dim3(8, 8, 8)>>>(r->accessSurface(), v->accessSurface(), f->accessSurface(), tn);
        restrict_kernel<<<dim3((tn/2 + 7) / 8, (tn/2 + 7) / 8, (tn/2 + 7) / 8), dim3(8, 8, 8)>>>(r2->accessSurface(), r->accessSurface(), tn/2);
        fillzero_kernel<<<dim3((tn/2 + 7) / 8, (tn/2 + 7) / 8, (tn/2 + 7) / 8), dim3(8, 8, 8)>>>(e2->accessSurface(), tn/2);
        vcycle(lev + 1, e2, r2);
        prolongate_kernel<<<dim3((tn/2 + 7) / 8, (tn/2 + 7) / 8, (tn/2 + 7) / 8), dim3(8, 8, 8)>>>(v->accessSurface(), e2->accessSurface(), tn/2);
        smooth(v, f, lev);
    }

    void projection() {
        divergence_kernel<<<dim3((n + 7) / 8, (n + 7) / 8, (n + 7) / 8), dim3(8, 8, 8)>>>(vel->accessSurface(), div->accessSurface(), n);
        vcycle(0, pre.get(), div.get());
        subgradient_kernel<<<dim3((n + 7) / 8, (n + 7) / 8, (n + 7) / 8), dim3(8, 8, 8)>>>(pre->accessSurface(), vel->accessSurface(), n);
    }

    void advection() {
        advect_kernel<<<dim3((n + 7) / 8, (n + 7) / 8, (n + 7) / 8), dim3(8, 8, 8)>>>(vel->accessTexture(), loc->accessSurface(), n);
        resample_kernel<<<dim3((n + 7) / 8, (n + 7) / 8, (n + 7) / 8), dim3(8, 8, 8)>>>(loc->accessSurface(), clr->accessTexture(), clrNext->accessSurface(), n);
        resample_kernel<<<dim3((n + 7) / 8, (n + 7) / 8, (n + 7) / 8), dim3(8, 8, 8)>>>(loc->accessSurface(), vel->accessTexture(), velNext->accessSurface(), n);

        std::swap(vel, velNext);
        std::swap(clr, clrNext);
    }

    void step(int times = 16) {
        for (int step = 0; step < times; step++) {
            projection();
            advection();
        }
    }

    /*void projection(int times = 400) {
        divergence_kernel<<<dim3((n + 7) / 8, (n + 7) / 8, (n + 7) / 8), dim3(8, 8, 8)>>>(vel.suf.access(), div.suf.access(), n);

        for (int step = 0; step < times; step++) {
            jacobi_kernel<<<dim3((n + 7) / 8, (n + 7) / 8, (n + 7) / 8), dim3(8, 8, 8)>>>(div.suf.access(), pre.suf.access(), preNext.suf.access(), n);
            std::swap(pre, preNext);
        }

        subgradient_kernel<<<dim3((n + 7) / 8, (n + 7) / 8, (n + 7) / 8), dim3(8, 8, 8)>>>(pre.suf.access(), vel.suf.access(), n);
    }*/

    float calc_loss() {
        divergence_kernel<<<dim3((n + 7) / 8, (n + 7) / 8, (n + 7) / 8), dim3(8, 8, 8)>>>(vel->accessSurface(), div->accessSurface(), n);
        float *sum;
        checkCudaErrors(hipMalloc(&sum, sizeof(float)));
        sumloss_kernel<<<dim3((n + 7) / 8, (n + 7) / 8, (n + 7) / 8), dim3(8, 8, 8)>>>(div->accessSurface(), sum, n);
        float cpu;
        checkCudaErrors(hipMemcpy(&cpu, sum, sizeof(float), hipMemcpyDeviceToHost));
        checkCudaErrors(hipFree(sum));
        return cpu;
    }
};

int main() {
    unsigned int n = 128;
    SmokeSim sim(n);

    {
        std::vector<float4> cpu(n * n * n);
        for (int z = 0; z < n; z++) {
            for (int y = 0; y < n; y++) {
                for (int x = 0; x < n; x++) {
                    float den = std::hypot(x - (int)n / 2, y - (int)n / 2, z - (int)n / 2) < n / 6 ? 1.f : 0.f;
                    cpu[x + n * (y + n * z)] = make_float4(den, 0.f, 0.f, 0.f);
                }
            }
        }
        sim.clr->copyIn(cpu.data());
    }

    {
        std::vector<float4> cpu(n * n * n);
        for (int z = 0; z < n; z++) {
            for (int y = 0; y < n; y++) {
                for (int x = 0; x < n; x++) {
                    float vel = std::hypot(x - (int)n / 2, y - (int)n / 2, z - (int)n / 2) < n / 6 ? 0.9f : 0.f;
                    cpu[x + n * (y + n * z)] = make_float4(0.f, 0.f, vel, 0.f);
                }
            }
        }
        sim.vel->copyIn(cpu.data());
    }

    std::vector<std::thread> tpool;
    for (int frame = 1; frame <= 250; frame++) {
        std::vector<float4> cpu(n * n * n);
        sim.clr->copyOut(cpu.data());
        tpool.push_back(std::thread([cpu = std::move(cpu), frame, n] {
            writevdb<float, 1>("/tmp/a" + std::to_string(1000 + frame).substr(1) + ".vdb", cpu.data(), n, n, n, sizeof(float4));
        }));

        printf("frame=%d, loss=%f\n", frame, sim.calc_loss());
        sim.step();
    }

    for (auto &t: tpool) t.join();
    return 0;
}
