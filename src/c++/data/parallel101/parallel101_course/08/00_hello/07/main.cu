#include <cstdio>
#include <hip/hip_runtime.h>

__host__ __device__ void say_hello() {
    printf("Hello, world!\n");
}

__global__ void kernel() {
    say_hello();
}

int main() {
    kernel<<<1, 1>>>();
    hipDeviceSynchronize();
    say_hello();
    return 0;
}
