#include <cstdio>
#include <hip/hip_runtime.h>

__global__ void kernel() {
    printf("Hello, world!\n");
}

int main() {
    kernel<<<1, 1>>>();
    hipDeviceSynchronize();
    return 0;
}
