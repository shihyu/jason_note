#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>

__global__ int kernel() {
    return 42;
}

int main() {
    int ret = kernel<<<1, 1>>>();
    hipDeviceSynchronize();
    printf("%d\n", ret);
    return 0;
}
