#include <cstdio>
#include <hip/hip_runtime.h>

__global__ void kernel(int *pret) {
    *pret = 42;
}

int main() {
    int ret = 0;
    kernel<<<1, 1>>>(&ret);
    hipDeviceSynchronize();
    printf("%d\n", ret);
    return 0;
}
