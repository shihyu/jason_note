#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"

__global__ void kernel(int *pret) {
    *pret = 42;
}

int main() {
    int *pret;
    checkCudaErrors(hipMalloc(&pret, sizeof(int)));
    kernel<<<1, 1>>>(pret);

    int ret;
    checkCudaErrors(hipMemcpy(&ret, pret, sizeof(int), hipMemcpyDeviceToHost));
    printf("result: %d\n", ret);

    hipFree(pret);
    return 0;
}
