#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"

__global__ void kernel(int *pret) {
    *pret = 42;
}

int main() {
    int *pret;
    checkCudaErrors(hipMalloc(&pret, sizeof(int)));
    kernel<<<1, 1>>>(pret);
    checkCudaErrors(hipDeviceSynchronize());
    hipFree(pret);
    return 0;
}
