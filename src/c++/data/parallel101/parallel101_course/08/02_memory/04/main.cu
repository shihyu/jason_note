#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"

__global__ void kernel(int *pret) {
    *pret = 42;
}

int main() {
    int ret = 0;
    kernel<<<1, 1>>>(&ret);
    checkCudaErrors(hipDeviceSynchronize());
    return 0;
}
