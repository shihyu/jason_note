#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#include <vector>
#include "CudaAllocator.h"
#include "ticktock.h"

__device__ __inline__ int my_atomic_add(int *dst, int src) {
    int old = *dst, expect;
    do {
        expect = old;
        old = atomicCAS(dst, expect, expect + src);
    } while (expect != old);
    return old;
}

__global__ void parallel_sum(int *sum, int const *arr, int n) {
    int local_sum = 0;
    for (int i = blockDim.x * blockIdx.x + threadIdx.x;
         i < n; i += blockDim.x * gridDim.x) {
        local_sum += arr[i];
    }
    my_atomic_add(&sum[0], local_sum);
}

int main() {
    int n = 65536;
    std::vector<int, CudaAllocator<int>> arr(n);
    std::vector<int, CudaAllocator<int>> sum(1);

    for (int i = 0; i < n; i++) {
        arr[i] = std::rand() % 4;
    }

    TICK(parallel_sum);
    parallel_sum<<<n / 4096, 512>>>(sum.data(), arr.data(), n);
    checkCudaErrors(hipDeviceSynchronize());
    TOCK(parallel_sum);

    printf("result: %d\n", sum[0]);

    return 0;
}
