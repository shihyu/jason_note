#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#include <vector>
#include "CudaAllocator.h"
#include "ticktock.h"

__global__ void parallel_sum(int *sum, int const *arr, int n) {
    for (int i = blockDim.x * blockIdx.x + threadIdx.x;
         i < n / 1024; i += blockDim.x * gridDim.x) {
        int local_sum[1024];
        for (int j = 0; j < 1024; j++) {
            local_sum[j] = arr[i * 1024 + j];
        }
        for (int j = 0; j < 512; j++) {
            local_sum[j] += local_sum[j + 512];
        }
        for (int j = 0; j < 256; j++) {
            local_sum[j] += local_sum[j + 256];
        }
        for (int j = 0; j < 128; j++) {
            local_sum[j] += local_sum[j + 128];
        }
        for (int j = 0; j < 64; j++) {
            local_sum[j] += local_sum[j + 64];
        }
        for (int j = 0; j < 32; j++) {
            local_sum[j] += local_sum[j + 32];
        }
        for (int j = 0; j < 16; j++) {
            local_sum[j] += local_sum[j + 16];
        }
        for (int j = 0; j < 8; j++) {
            local_sum[j] += local_sum[j + 8];
        }
        for (int j = 0; j < 4; j++) {
            local_sum[j] += local_sum[j + 4];
        }
        for (int j = 0; j < 2; j++) {
            local_sum[j] += local_sum[j + 2];
        }
        for (int j = 0; j < 1; j++) {
            local_sum[j] += local_sum[j + 1];
        }
        sum[i] = local_sum[0];
    }
}

int main() {
    int n = 1<<24;
    std::vector<int, CudaAllocator<int>> arr(n);
    std::vector<int, CudaAllocator<int>> sum(n / 1024);

    for (int i = 0; i < n; i++) {
        arr[i] = std::rand() % 4;
    }

    TICK(parallel_sum);
    parallel_sum<<<n / 1024 / 128, 128>>>(sum.data(), arr.data(), n);
    checkCudaErrors(hipDeviceSynchronize());

    int final_sum = 0;
    for (int i = 0; i < n / 1024; i++) {
        final_sum += sum[i];
    }
    TOCK(parallel_sum);

    printf("result: %d\n", final_sum);

    return 0;
}
