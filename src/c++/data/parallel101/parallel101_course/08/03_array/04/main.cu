#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"

__global__ void kernel(int *arr, int n) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    arr[i] = i;
}

int main() {
    int n = 65536;
    int *arr;
    checkCudaErrors(hipMallocManaged(&arr, n * sizeof(int)));

    int nthreads = 128;
    int nblocks = n / nthreads;
    kernel<<<nblocks, nthreads>>>(arr, n);

    checkCudaErrors(hipDeviceSynchronize());
    for (int i = 0; i < n; i++) {
        printf("arr[%d]: %d\n", i, arr[i]);
    }

    hipFree(arr);
    return 0;
}
