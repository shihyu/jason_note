#include <cstdio>
#include <hip/hip_runtime.h>

__global__ void kernel() {
    printf("Hello, world!\n");
}

int main() {
    kernel<<<1, 3>>>();
    hipDeviceSynchronize();
    return 0;
}
