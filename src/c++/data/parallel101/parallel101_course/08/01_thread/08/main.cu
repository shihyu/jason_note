#include <cstdio>
#include <hip/hip_runtime.h>

__global__ void kernel() {
    printf("Block (%d,%d) of (%d,%d), Thread (%d,%d) of (%d,%d)\n",
           blockIdx.x, blockIdx.y,
           gridDim.x, gridDim.y,
           threadIdx.x, threadIdx.y,
           blockDim.x, blockDim.y);
}

int main() {
    kernel<<<dim3(2, 1, 1), dim3(3, 2, 1)>>>();
    hipDeviceSynchronize();
    return 0;
}
