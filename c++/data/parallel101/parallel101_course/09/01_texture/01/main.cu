#include "hip/hip_runtime.h"
#include <cstdio>
#include <vector>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#include "hip/hip_vector_types.h"
#include "CudaArray.cuh"
#include "ticktock.h"
#include "writevdb.h"

__global__ void advect_kernel(CudaTexture<float4>::Accessor texVel, CudaSurface<float4>::Accessor sufLoc, unsigned int n) {
    unsigned int x = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int y = threadIdx.y + blockDim.y * blockIdx.y;
    unsigned int z = threadIdx.z + blockDim.z * blockIdx.z;
    if (x >= n || y >= n || z >= n) return;

    float4 vel = texVel.sample(x, y, z);
    float4 loc = make_float4(x + 0.5f, y + 0.5f, z + 0.5f, 42.f) - vel;
    sufLoc.write(loc, x, y, z);
}

__global__ void resample_kernel(CudaSurface<float4>::Accessor sufLoc, CudaTexture<float4>::Accessor texClr, CudaSurface<float4>::Accessor sufClrNext, unsigned int n) {
    unsigned int x = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int y = threadIdx.y + blockDim.y * blockIdx.y;
    unsigned int z = threadIdx.z + blockDim.z * blockIdx.z;
    if (x >= n || y >= n || z >= n) return;

    float4 loc = sufLoc.read(x, y, z);
    float4 clr = texClr.sample(loc.x, loc.y, loc.z);
    sufClrNext.write(clr, x, y, z);
}

__global__ void divergence_kernel(CudaSurface<float4>::Accessor sufVel, CudaSurface<float>::Accessor sufDiv, unsigned int n) {
    unsigned int x = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int y = threadIdx.y + blockDim.y * blockIdx.y;
    unsigned int z = threadIdx.z + blockDim.z * blockIdx.z;
    if (x >= n || y >= n || z >= n) return;

    float vxp = sufVel.read<hipBoundaryModeClamp>(x + 1, y, z).x;
    float vxn = sufVel.read<hipBoundaryModeClamp>(x - 1, y, z).x;
    float vyp = sufVel.read<hipBoundaryModeClamp>(x, y + 1, z).y;
    float vyn = sufVel.read<hipBoundaryModeClamp>(x, y - 1, z).y;
    float vzp = sufVel.read<hipBoundaryModeClamp>(x, y, z + 1).z;
    float vzn = sufVel.read<hipBoundaryModeClamp>(x, y, z - 1).z;
    float div = (vxp - vxn + vyp - vyn + vzp - vzn) * 0.5f;
    sufDiv.write(div, x, y, z);
}

__global__ void sumloss_kernel(CudaSurface<float>::Accessor sufDiv, float *sum, unsigned int n) {
    unsigned int x = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int y = threadIdx.y + blockDim.y * blockIdx.y;
    unsigned int z = threadIdx.z + blockDim.z * blockIdx.z;
    if (x >= n || y >= n || z >= n) return;

    float div = sufDiv.read(x, y, z);
    atomicAdd(sum, div * div);
}

__global__ void jacobi_kernel(CudaSurface<float>::Accessor sufDiv, CudaSurface<float>::Accessor sufPre, CudaSurface<float>::Accessor sufPreNext, unsigned int n) {
    unsigned int x = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int y = threadIdx.y + blockDim.y * blockIdx.y;
    unsigned int z = threadIdx.z + blockDim.z * blockIdx.z;
    if (x >= n || y >= n || z >= n) return;

    float pxp = sufPre.read<hipBoundaryModeClamp>(x + 1, y, z);
    float pxn = sufPre.read<hipBoundaryModeClamp>(x - 1, y, z);
    float pyp = sufPre.read<hipBoundaryModeClamp>(x, y + 1, z);
    float pyn = sufPre.read<hipBoundaryModeClamp>(x, y - 1, z);
    float pzp = sufPre.read<hipBoundaryModeClamp>(x, y, z + 1);
    float pzn = sufPre.read<hipBoundaryModeClamp>(x, y, z - 1);
    float div = sufDiv.read(x, y, z);
    float preNext = (pxp + pxn + pyp + pyn + pzp + pzn - div) * (1.f / 6.f);
    sufPreNext.write(preNext, x, y, z);
}

__global__ void subgradient_kernel(CudaSurface<float>::Accessor sufPre, CudaSurface<float4>::Accessor sufVel, unsigned int n) {
    unsigned int x = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int y = threadIdx.y + blockDim.y * blockIdx.y;
    unsigned int z = threadIdx.z + blockDim.z * blockIdx.z;
    if (x >= n || y >= n || z >= n) return;

    float pxp = sufPre.read<hipBoundaryModeClamp>(x + 1, y, z);
    float pxn = sufPre.read<hipBoundaryModeClamp>(x - 1, y, z);
    float pyp = sufPre.read<hipBoundaryModeClamp>(x, y + 1, z);
    float pyn = sufPre.read<hipBoundaryModeClamp>(x, y - 1, z);
    float pzp = sufPre.read<hipBoundaryModeClamp>(x, y, z + 1);
    float pzn = sufPre.read<hipBoundaryModeClamp>(x, y, z - 1);
    float4 vel = sufVel.read(x, y, z);
    vel.x -= 0.5f * (pxp - pxn);
    vel.y -= 0.5f * (pyp - pyn);
    vel.z -= 0.5f * (pzp - pzn);
    sufVel.write(vel, x, y, z);
}

struct SmokeSim {
    nocopy_t nocopy;

    unsigned int n;
    CudaAST<float4> loc;
    CudaAST<float4> vel;
    CudaAST<float4> velNext;
    CudaAST<float4> clr;
    CudaAST<float4> clrNext;
    CudaAST<float> div;
    CudaAST<float> pre;
    CudaAST<float> preNext;

    SmokeSim(ctor_t, unsigned int _n)
    : n(_n)
    , loc(ctor, {{n, n, n}})
    , vel(ctor, {{n, n, n}})
    , velNext(ctor, {{n, n, n}})
    , clr(ctor, {{n, n, n}})
    , clrNext(ctor, {{n, n, n}})
    , div(ctor, {{n, n, n}})
    , pre(ctor, {{n, n, n}})
    , preNext(ctor, {{n, n, n}})
    {}

    void advection() {
        advect_kernel<<<dim3((n + 7) / 8, (n + 7) / 8, (n + 7) / 8), dim3(8, 8, 8)>>>(vel.tex.access(), loc.suf.access(), n);
        resample_kernel<<<dim3((n + 7) / 8, (n + 7) / 8, (n + 7) / 8), dim3(8, 8, 8)>>>(loc.suf.access(), clr.tex.access(), clrNext.suf.access(), n);
        resample_kernel<<<dim3((n + 7) / 8, (n + 7) / 8, (n + 7) / 8), dim3(8, 8, 8)>>>(loc.suf.access(), vel.tex.access(), velNext.suf.access(), n);

        std::swap(vel, velNext);
        std::swap(clr, clrNext);
    }

    void projection(int times = 400) {
        divergence_kernel<<<dim3((n + 7) / 8, (n + 7) / 8, (n + 7) / 8), dim3(8, 8, 8)>>>(vel.suf.access(), div.suf.access(), n);

        for (int step = 0; step < times; step++) {
            jacobi_kernel<<<dim3((n + 7) / 8, (n + 7) / 8, (n + 7) / 8), dim3(8, 8, 8)>>>(div.suf.access(), pre.suf.access(), preNext.suf.access(), n);
            std::swap(pre, preNext);
        }

        subgradient_kernel<<<dim3((n + 7) / 8, (n + 7) / 8, (n + 7) / 8), dim3(8, 8, 8)>>>(pre.suf.access(), vel.suf.access(), n);
    }

    float calc_loss() {
        divergence_kernel<<<dim3((n + 7) / 8, (n + 7) / 8, (n + 7) / 8), dim3(8, 8, 8)>>>(vel.suf.access(), div.suf.access(), n);
        float *sum;
        checkCudaErrors(hipMalloc(&sum, sizeof(float)));
        sumloss_kernel<<<dim3((n + 7) / 8, (n + 7) / 8, (n + 7) / 8), dim3(8, 8, 8)>>>(div.suf.access(), sum, n);
        float cpu;
        checkCudaErrors(hipMemcpy(&cpu, sum, sizeof(float), hipMemcpyDeviceToHost));
        checkCudaErrors(hipFree(sum));
        return cpu;
    }
};

int main() {
    unsigned int n = 64;
    SmokeSim sim(ctor, n);

    {
        std::vector<float4> cpu(n * n * n);
        for (unsigned int z = 0; z < n; z++) {
            for (unsigned int y = 0; y < n; y++) {
                for (unsigned int x = 0; x < n; x++) {
                    float den = std::hypot((int)x - (int)n / 2, (int)y - (int)n / 2, (int)z - (int)n / 2) < n / 3 ? 1.f : 0.f;
                    cpu[x + n * (y + n * z)] = make_float4(den, 0.f, 0.f, 0.f);
                }
            }
        }
        sim.clr.arr.copyIn(cpu.data());
    }

    {
        std::vector<float4> cpu(n * n * n);
        for (unsigned int z = 0; z < n; z++) {
            for (unsigned int y = 0; y < n; y++) {
                for (unsigned int x = 0; x < n; x++) {
                    float vel = std::hypot((int)x - (int)n / 2, (int)y - (int)n / 2, (int)z - (int)n / 2) < n / 3 ? 0.5f : 0.f;
                    cpu[x + n * (y + n * z)] = make_float4(vel, 0.f, 0.f, 0.f);
                }
            }
        }
        sim.vel.arr.copyIn(cpu.data());
    }

    std::vector<float4> cpu(n * n * n);
    for (int frame = 1; frame <= 100; frame++) {
        sim.clr.arr.copyOut(cpu.data());
        writevdb<float, 1>("/tmp/a" + std::to_string(1000 + frame).substr(1) + ".vdb", cpu.data(), n, n, n, sizeof(float4));

        printf("frame=%d, loss=%f\n", frame, sim.calc_loss());
        sim.advection();
        sim.projection();
    }

    return 0;
}
