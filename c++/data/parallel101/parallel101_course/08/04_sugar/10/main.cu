#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#include <vector>

template <class T>
struct CudaAllocator {
    using value_type = T;

    T *allocate(size_t size) {
        T *ptr = nullptr;
        checkCudaErrors(hipMallocManaged(&ptr, size * sizeof(T)));
        return ptr;
    }

    void deallocate(T *ptr, size_t size = 0) {
        checkCudaErrors(hipFree(ptr));
    }

    template <class ...Args>
    void construct(T *p, Args &&...args) {
        if constexpr (!(sizeof...(Args) == 0 && std::is_pod_v<T>))
            ::new((void *)p) T(std::forward<Args>(args)...);
    }
};

template <class Func>
__global__ void parallel_for(int n, Func func) {
    for (int i = blockDim.x * blockIdx.x + threadIdx.x;
         i < n; i += blockDim.x * gridDim.x) {
        func(i);
    }
}

int main() {
    int n = 65536;
    std::vector<int, CudaAllocator<int>> arr(n);

    parallel_for<<<32, 128>>>(n, [arr = arr.data()] __device__ (int i) {
        arr[i] = i;
    });

    checkCudaErrors(hipDeviceSynchronize());
    for (int i = 0; i < n; i++) {
        printf("arr[%d] = %d\n", i, arr[i]);
    }

    return 0;
}
