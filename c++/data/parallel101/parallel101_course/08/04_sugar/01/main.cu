#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#include <vector>

template <class T>
struct CudaAllocator {
    using value_type = T;

    T *allocate(size_t size) {
        T *ptr = nullptr;
        checkCudaErrors(hipMallocManaged(&ptr, size * sizeof(T)));
        return ptr;
    }

    void deallocate(T *ptr, size_t size = 0) {
        checkCudaErrors(hipFree(ptr));
    }
};

__global__ void kernel(int *arr, int n) {
    for (int i = blockDim.x * blockIdx.x + threadIdx.x;
         i < n; i += blockDim.x * gridDim.x) {
        arr[i] = i;
    }
}

int main() {
    int n = 65536;
    std::vector<int, CudaAllocator<int>> arr(n);

    kernel<<<32, 128>>>(arr.data(), n);

    checkCudaErrors(hipDeviceSynchronize());
    for (int i = 0; i < n; i++) {
        printf("arr[%d]: %d\n", i, arr[i]);
    }

    return 0;
}
