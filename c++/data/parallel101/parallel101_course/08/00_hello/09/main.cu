#include <cstdio>
#include <hip/hip_runtime.h>

__host__ __device__ void say_hello() {
#ifdef __CUDA_ARCH__
    printf("Hello, world from GPU!\n");
#else
    printf("Hello, world from CPU!\n");
#endif
}

__global__ void kernel() {
    say_hello();
}

int main() {
    kernel<<<1, 1>>>();
    hipDeviceSynchronize();
    say_hello();
    return 0;
}
