#include <cstdio>
#include <hip/hip_runtime.h>

__device__ void say_hello() {
    printf("Hello, world from GPU!\n");
}

void say_hello_host() {
    printf("Hello, world from CPU!\n");
}

__global__ void kernel() {
    say_hello();
}

int main() {
    kernel<<<1, 1>>>();
    hipDeviceSynchronize();
    say_hello_host();
    return 0;
}
