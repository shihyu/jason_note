#include <cstdio>
#include <hip/hip_runtime.h>

__global__ void kernel() {
    printf("Thread %d of %d\n", threadIdx.x, blockDim.x);
}

int main() {
    kernel<<<2, 3>>>();
    hipDeviceSynchronize();
    return 0;
}
