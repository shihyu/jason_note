#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"

__global__ void kernel(int *pret) {
    *pret = 42;
}

int main() {
    int *pret;
    checkCudaErrors(hipMallocManaged(&pret, sizeof(int)));
    kernel<<<1, 1>>>(pret);
    checkCudaErrors(hipDeviceSynchronize());
    printf("result: %d\n", *pret);
    hipFree(pret);
    return 0;
}
