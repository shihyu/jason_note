#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"

__global__ void kernel(int *pret) {
    *pret = 42;
}

int main() {
    int *pret = (int *)malloc(sizeof(int));
    kernel<<<1, 1>>>(pret);
    checkCudaErrors(hipDeviceSynchronize());
    free(pret);
    return 0;
}
