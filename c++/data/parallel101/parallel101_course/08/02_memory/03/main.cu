#include <cstdio>
#include <hip/hip_runtime.h>

__global__ void kernel(int *pret) {
    *pret = 42;
}

int main() {
    int ret = 0;
    kernel<<<1, 1>>>(&ret);
    hipError_t err = hipDeviceSynchronize();
    printf("error code: %d\n", err);
    printf("error name: %s\n", hipGetErrorName(err));
    printf("%d\n", ret);
    return 0;
}
