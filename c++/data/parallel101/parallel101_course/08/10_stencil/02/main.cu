#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#include <vector>
#include "CudaAllocator.h"
#include "ticktock.h"
#include "stb_image.h"
#include "stb_image_write.h"

template <class A>
std::tuple<int, int, int> read_image(A &a, const char *path) {
    int nx = 0, ny = 0, comp = 0;
    unsigned char *p = stbi_load(path, &nx, &ny, &comp, 0);
    if (!p) {
        perror(path);
        exit(-1);
    }
    a.resize(nx * ny * comp);
    for (int c = 0; c < comp; c++) {
        for (int y = 0; y < ny; y++) {
            for (int x = 0; x < nx; x++) {
                a[c * nx * ny + y * nx + x] = (1.f / 255.f) * p[(y * nx + x) * comp + c];
            }
        }
    }
    stbi_image_free(p);
    return {nx, ny, comp};
}

template <class A>
void write_image(A const &a, int nx, int ny, int comp, const char *path) {
    auto p = (unsigned char *)malloc(nx * ny * comp);
    for (int c = 0; c < comp; c++) {
        for (int y = 0; y < ny; y++) {
            for (int x = 0; x < nx; x++) {
                p[(y * nx + x) * comp + c] = std::max(0.f, std::min(255.f, a[c * nx * ny + y * nx + x] * 255.f));
            }
        }
    }
    int ret = 0;
    auto pt = strrchr(path, '.');
    if (pt && !strcmp(pt, ".png")) {
        ret = stbi_write_png(path, nx, ny, comp, p, 0);
    } else if (pt && !strcmp(pt, ".jpg")) {
        ret = stbi_write_jpg(path, nx, ny, comp, p, 0);
    } else {
        ret = stbi_write_bmp(path, nx, ny, comp, p);
    }
    free(p);
    if (!ret) {
        perror(path);
        exit(-1);
    }
}

template <int nblur, int blockSize>
__global__ void parallel_yblur(float *out, float const *in, int nx, int ny) {
    int x = blockIdx.x * blockSize + threadIdx.x;
    int y = blockIdx.y * blockSize + threadIdx.y;
    if (x >= nx || y >= ny) return;
    float sum = 0;
    for (int i = 0; i < nblur; i++) {
        sum += in[std::min(y + i, ny - 1) * nx + x];
    }
    out[y * nx + x] = sum / nblur;
}

int main() {
    std::vector<float, CudaAllocator<float>> in;
    std::vector<float, CudaAllocator<float>> out;

    auto [nx, ny, comp] = read_image(in, "original.jpg");
    out.resize(in.size());

    TICK(parallel_yblur);
    parallel_yblur<32, 32><<<dim3((nx + 31) / 32, (ny + 31) / 32, 1), dim3(32, 32, 1)>>>
        (out.data(), in.data(), nx, ny);
    checkCudaErrors(hipDeviceSynchronize());
    TOCK(parallel_yblur);

    write_image(out, nx, ny, 1, "/tmp/out.png");
    system("display /tmp/out.png &");
    return 0;
}
