#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#include <vector>
#include "CudaAllocator.h"
#include "ticktock.h"
#include "stb_image.h"
#include "stb_image_write.h"

template <class A>
std::tuple<int, int, int> read_image(A &a, const char *path) {
    int nx = 0, ny = 0, comp = 0;
    unsigned char *p = stbi_load(path, &nx, &ny, &comp, 0);
    if (!p) {
        perror(path);
        exit(-1);
    }
    a.resize(nx * ny * comp);
    for (int c = 0; c < comp; c++) {
        for (int y = 0; y < ny; y++) {
            for (int x = 0; x < nx; x++) {
                a[c * nx * ny + y * nx + x] = (1.f / 255.f) * p[(y * nx + x) * comp + c];
            }
        }
    }
    stbi_image_free(p);
    return {nx, ny, comp};
}

template <class A>
void write_image(A const &a, int nx, int ny, int comp, const char *path) {
    auto p = (unsigned char *)malloc(nx * ny * comp);
    for (int c = 0; c < comp; c++) {
        for (int y = 0; y < ny; y++) {
            for (int x = 0; x < nx; x++) {
                p[(y * nx + x) * comp + c] = std::max(0.f, std::min(255.f, a[c * nx * ny + y * nx + x] * 255.f));
            }
        }
    }
    int ret = 0;
    auto pt = strrchr(path, '.');
    if (pt && !strcmp(pt, ".png")) {
        ret = stbi_write_png(path, nx, ny, comp, p, 0);
    } else if (pt && !strcmp(pt, ".jpg")) {
        ret = stbi_write_jpg(path, nx, ny, comp, p, 0);
    } else {
        ret = stbi_write_bmp(path, nx, ny, comp, p);
    }
    free(p);
    if (!ret) {
        perror(path);
        exit(-1);
    }
}

template <int iters, int blockSize>
__global__ void parallel_jacobi_kernel(float *out, float const *in, int nx, int ny) {
    int blockX = blockIdx.x;
    int blockY = blockIdx.y;
    int threadX = threadIdx.x;
    int threadY = threadIdx.y;
    constexpr int chunkSize = blockSize - iters * 2;
    int globalX = blockX * chunkSize - iters + threadX;
    int globalY = blockY * chunkSize - iters + threadY;

    __shared__ float mem[2][blockSize + 2][blockSize + 2];
    int clampedX = std::min(std::max(globalX, 0), nx - 1);
    int clampedY = std::min(std::max(globalY, 0), ny - 1);
    mem[0][1 + threadY][1 + threadX] = in[nx * clampedY + clampedX]; 

    if (threadY == 0) {
        int clampedYn = std::min(std::max(blockY * chunkSize - iters - 1, 0), ny - 1);
        mem[0][0][1 + threadX] = in[nx * clampedYn + clampedX]; 
        int clampedYp = std::min(std::max(blockY * chunkSize - iters + blockSize, 0), ny - 1);
        mem[0][1 + blockSize][1 + threadX] = in[nx * clampedYp + clampedX]; 
    }

    if (threadX == 0) {
        int clampedXn = std::min(std::max(blockX * chunkSize - iters - 1, 0), nx - 1);
        mem[0][1 + threadY][0] = in[nx * clampedY + clampedXn]; 
        int clampedXp = std::min(std::max(blockX * chunkSize - iters + blockSize, 0), nx - 1);
        mem[0][1 + threadY][1 + blockSize] = in[nx * clampedY + clampedXp]; 
    }

    __syncthreads();

    for (int stage = 0; stage < iters; stage += 2) {
#pragma unroll
        for (int phase = 0; phase < 2; phase++) {
            mem[1 - phase][1 + threadY][1 + threadX] =
                ( mem[phase][1 + threadY + 1][1 + threadX]
                + mem[phase][1 + threadY - 1][1 + threadX]
                + mem[phase][1 + threadY][1 + threadX + 1]
                + mem[phase][1 + threadY][1 + threadX - 1]
                ) / 4;
            __syncthreads();
        }
    }

    if (threadX >= iters && threadX < blockSize - iters)
        if (threadY >= iters && threadY < blockSize - iters)
            if (globalX < nx && globalY < ny)
                out[globalY * nx + globalX] = mem[0][1 + threadY][1 + threadX];
}

template <int iters, int blockSize>
void parallel_jacobi(float *out, float const *in, int nx, int ny) {
    constexpr int chunkSize = blockSize - iters * 2;
    static_assert(chunkSize > 0 && iters % 2 == 0);
    parallel_jacobi_kernel<iters, blockSize>
        <<<dim3((nx + chunkSize - 1) / chunkSize, (ny + chunkSize - 1) / chunkSize, 1), 
        dim3(blockSize, blockSize, 1)>>>(out, in, nx, ny);
}

int main() {
    std::vector<float, CudaAllocator<float>> in;
    std::vector<float, CudaAllocator<float>> out;

    auto [nx, ny, comp] = read_image(in, "original.jpg");
    out.resize(in.size());

    TICK(parallel_jacobi);

    constexpr int iters = 4;
    for (int step = 0; step < 1024; step += iters) {
        parallel_jacobi<iters, 32>(out.data(), in.data(), nx, ny);
        std::swap(out, in);
    }

    checkCudaErrors(hipDeviceSynchronize());
    TOCK(parallel_jacobi);

    write_image(in, nx, ny, 1, "/tmp/out.png");
    system("display /tmp/out.png &");
    return 0;
}
