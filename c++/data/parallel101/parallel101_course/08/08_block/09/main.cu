#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#include <vector>
#include "CudaAllocator.h"
#include "ticktock.h"

template <int blockSize, class T>
__global__ void parallel_sum_kernel(T *sum, T const *arr, int n) {
    __shared__ volatile int local_sum[blockSize];
    int j = threadIdx.x;
    int i = blockIdx.x;
    T temp_sum = 0;
    for (int t = i * blockSize + j; t < n; t += blockSize * gridDim.x) {
        temp_sum += arr[t];
    }
    local_sum[j] = temp_sum;
    __syncthreads();
    if constexpr (blockSize >= 1024) {
        if (j < 512)
            local_sum[j] += local_sum[j + 512];
        __syncthreads();
    }
    if constexpr (blockSize >= 512) {
        if (j < 256)
            local_sum[j] += local_sum[j + 256];
        __syncthreads();
    }
    if constexpr (blockSize >= 256) {
        if (j < 128)
            local_sum[j] += local_sum[j + 128];
        __syncthreads();
    }
    if constexpr (blockSize >= 128) {
        if (j < 64)
            local_sum[j] += local_sum[j + 64];
        __syncthreads();
    }
    if (j < 32) {
        if constexpr (blockSize >= 64)
            local_sum[j] += local_sum[j + 32];
        if constexpr (blockSize >= 32)
            local_sum[j] += local_sum[j + 16];
        if constexpr (blockSize >= 16)
            local_sum[j] += local_sum[j + 8];
        if constexpr (blockSize >= 8)
            local_sum[j] += local_sum[j + 4];
        if constexpr (blockSize >= 4)
            local_sum[j] += local_sum[j + 2];
        if (j == 0) {
            sum[i] = local_sum[0] + local_sum[1];
        }
    }
}

template <int reduceScale = 4096, int blockSize = 256, class T>
int parallel_sum(T const *arr, int n) {
    std::vector<int, CudaAllocator<int>> sum(n / reduceScale);
    parallel_sum_kernel<blockSize><<<n / reduceScale, blockSize>>>(sum.data(), arr, n);
    checkCudaErrors(hipDeviceSynchronize());
    T final_sum = 0;
    for (int i = 0; i < n / reduceScale; i++) {
        final_sum += sum[i];
    }
    return final_sum;
}

int main() {
    int n = 1<<24;
    std::vector<int, CudaAllocator<int>> arr(n);
    std::vector<int, CudaAllocator<int>> sum(n / 4096);

    for (int i = 0; i < n; i++) {
        arr[i] = std::rand() % 4;
    }

    TICK(parallel_sum);
    int final_sum = parallel_sum(arr.data(), n);
    TOCK(parallel_sum);

    printf("result: %d\n", final_sum);

    return 0;
}
