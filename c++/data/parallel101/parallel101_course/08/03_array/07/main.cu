#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"

__global__ void kernel(int *arr, int n) {
    for (int i = blockDim.x * blockIdx.x + threadIdx.x;
         i < n; i += blockDim.x * gridDim.x) {
        arr[i] = i;
    }
}

int main() {
    int n = 65536;
    int *arr;
    checkCudaErrors(hipMallocManaged(&arr, n * sizeof(int)));

    kernel<<<32, 128>>>(arr, n);

    checkCudaErrors(hipDeviceSynchronize());
    for (int i = 0; i < n; i++) {
        printf("arr[%d]: %d\n", i, arr[i]);
    }

    hipFree(arr);
    return 0;
}
