#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"

__global__ void kernel(int *arr, int n) {
    for (int i = 0; i < n; i++) {
        arr[i] = i;
    }
}

int main() {
    int n = 32;
    int *arr;
    checkCudaErrors(hipMallocManaged(&arr, n * sizeof(int)));
    kernel<<<1, 1>>>(arr, n);
    checkCudaErrors(hipDeviceSynchronize());
    for (int i = 0; i < n; i++) {
        printf("arr[%d]: %d\n", i, arr[i]);
    }
    hipFree(arr);
    return 0;
}
