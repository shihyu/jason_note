#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#include <vector>
#include "CudaAllocator.h"
#include "ticktock.h"

__global__ void parallel_sum(int *sum, int const *arr, int n) {
    for (int i = blockDim.x * blockIdx.x + threadIdx.x;
         i < n; i += blockDim.x * gridDim.x) {
        atomicAdd(&sum[0], arr[i]);
    }
}

int main() {
    int n = 65536;
    std::vector<int, CudaAllocator<int>> arr(n);
    std::vector<int, CudaAllocator<int>> sum(1);

    for (int i = 0; i < n; i++) {
        arr[i] = std::rand() % 4;
    }

    TICK(parallel_sum);
    parallel_sum<<<n / 128, 128>>>(sum.data(), arr.data(), n);
    checkCudaErrors(hipDeviceSynchronize());
    TOCK(parallel_sum);

    printf("result: %d\n", sum[0]);

    return 0;
}
